#include "hip/hip_runtime.h"
#include "kernel.h"

#define TX 32 // number of threads along x-axis
#define TY 32 // number of threads along y-axis
#define RAD 1

int divUp(int a, int b) {
	return (a + b - 1) / b; // number of blocks of a specified size to cover a computational grid
}

__device__ int idxClip(int idx, int idxMax) {
	return idx > (idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx); // make sure indexes are between (0, N - 1)
}

__device__ int flatten(int col, int row, int width, int height) {
	return idxClip(col, width) + idxClip(row, height) * width; // flatten a 2D array to 1D	
}

__device__ unsigned char clip(int n) {
	return n > 255 ? 255 : (n < 0 ? 0 : n);
}

__global__ void resetKernel(float *d_temp, int w, int h, BC bc) {
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;

	if((col >= w) || (row >= h)) return;
	d_temp[row * w + col] = bc.t_a;
}

__global__ void tempKernel(uchar4 *d_out, float *d_temp, int w, int h, BC bc) {
	extern __shared__ float s_in[];

	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	if((col >= w) || (row >= h)) return;

	const int idx = flatten(col, row, w, h);

	// local height and width
	const int s_w = blockDim.x + 2 * RAD;
	const int s_h = blockDim.y + 2 * RAD;

	// shared memory indices
	const int s_col = threadIdx.x + RAD;
	const int s_row = threadIdx.y + RAD;
	const int s_idx = flatten(s_col, s_row, s_w, s_h); // flatten the shared memory

	d_out[idx].x = 0;
	d_out[idx].y = 0;
	d_out[idx].z = 0;
	d_out[idx].w = 255;

	// let's play

	// load regular cells
	s_in[s_idx] = d_temp[idx];
	
	if(threadIdx.x < RAD) {
		// load column neighbors
		s_in[flatten(s_col - RAD, s_row, s_w, s_h)] = d_temp[flatten(col - RAD, row, w, h)];
		s_in[flatten(s_col + blockDim.x, s_row, s_w, s_h)] = d_temp[flatten(col + blockDim.x, row, w, h)];
	}

	if(threadIdx.y < RAD) {
		// load row neighbors
		s_in[flatten(s_col, s_row - RAD, s_w, s_h)] = d_temp[flatten(col, row - RAD, w, h)];
		s_in[flatten(s_col, s_row + blockDim.y, s_w, s_h)] = d_temp[flatten(col, row + blockDim.y, w, h)];
	}

	float dSq = ((col - bc.x) * (col - bc.x)) + (row - bc.y) * (row - bc.y);

	// if inside the pipe, set temp to source temperature and return

	if(dSq < bc.rad * bc.rad) {
		d_temp[idx] = bc.t_s;
		return;
	}

	// if outside the plate, set the temp to ambient
	if((col == 0) || (col == w - 1) || (row == 0) || (col + row < 0) || (col - row > w)) {
		d_temp[idx] = bc.t_a;
		return;
	}

	// if point is below ground, set temp to ground
	if(row == h - 1) {
		d_temp[idx] = bc.t_g;
		return;
	}

	__syncthreads();

	// for all remaining points, find temperatures and set colors

	float temp = 0.25f * (s_in[flatten(s_col - 1, s_row, s_w, s_h)] + s_in[flatten(s_col + 1, s_row, s_w, s_h)] + s_in[flatten(s_col, s_row - 1, s_w, s_h)] + s_in[flatten(s_col, s_row + 1, s_w, s_h)]);
	d_temp[idx] = temp;
	const unsigned char intensity = clip((int) temp);
	d_out[idx].x = intensity; // more heat -> more read
	d_out[idx].z = 255 - intensity; // less heat -> more blue
}

void kernelLauncher(uchar4 *d_out, float *d_temp, int w, int h, BC bc) {
	const dim3 blockSize(TX, TY);
	const dim3 gridSize(divUp(w, TX), divUp(h, TY));
	const size_t smSz = (TX + 2 * RAD) * (TY + 2 * RAD) * sizeof(float); // shared memory size
	tempKernel<<<gridSize, blockSize, smSz>>>(d_out, d_temp, w, h ,bc);
}

void resetTemperature(float *d_temp, int w, int h, BC bc) {
	const dim3 blockSize(TX, TY);
	const dim3 gridSize(divUp(w, TX), divUp(h, TY));
	resetKernel<<<gridSize, blockSize>>>(d_temp, w, h, bc);
}